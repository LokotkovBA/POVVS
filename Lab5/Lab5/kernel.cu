#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "targ.h"
#include <stdio.h>
#include <cstdint>


__global__ void bilinear_interpolation_kernel(uint8_t *output, uint8_t *input, uint8_t pitchOutput, uint8_t pitchInput, uint8_t bytesPerPixelInput, uint8_t bytesPerPixelOutput, float xRatio, float yRatio) {
	int x = (int)(xRatio * blockIdx.x);
	int y = (int)(yRatio * blockIdx.y);

	uint8_t *a; uint8_t *b; uint8_t *c; uint8_t *d;
	float xDist, yDist, blue, red, green;

	// X and Y distance difference
	xDist = (xRatio * blockIdx.x) - x;
	yDist = (yRatio * blockIdx.y) - y;

	// Points
	a = input + y * pitchInput + x * bytesPerPixelInput;
	b = input + y * pitchInput + (x + 1) * bytesPerPixelInput;
	c = input + (y + 1) * pitchInput + x * bytesPerPixelInput;
	d = input + (y + 1) * pitchInput + (x + 1) * bytesPerPixelInput;

	// Calc
	blue = (a[2])*(1 - xDist)*(1 - yDist) + (b[2])*(xDist)*(1 - yDist) + (c[2])*(yDist)*(1 - xDist) + (d[2])*(xDist * yDist);

	uint8_t *p = output + blockIdx.y * pitchOutput + blockIdx.x * bytesPerPixelOutput;
	*(uint32_t*)p = 0xff000000 | ((((int)red) << 16)) | ((((int)green) << 8)) | ((int)blue);
}


int main()
{
	unsigned char* data;


	unsigned width, height, pbpp;
	int blocks = 8;
	if (!Targa2Array("C:/Users/B.Lokotkov/Desktop/_git/POVVS/Lab5/sample2.tga", &data, &width, &height, &pbpp))
	{
		std::cout << "Can't read file";
		return -1;
	}
	int N, nblocks, nthreads;
	std::cout << "N = ";
	std::cin >> N;

    return 0;
}

/*// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}*/
