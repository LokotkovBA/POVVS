#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "targ.h"
#include <stdio.h>
#include <cstdint>
#include <conio.h>

#define N 5
unsigned char* data1;
unsigned width, height, pbpp;


__global__ void fun_kernel(unsigned char* result_data, unsigned char* data1, int height, int width, int steps)
{
	int thread = blockIdx.x * blockDim.x + threadIdx.x;

	int neww = (width - 1)*N + 1;
	int newh = (height - 1)*N + 1;
	float res, d1, d2, d3, d4, u, t;
	int h, w, p1, p2, p3, p4;

	float pom = (height - 1) / (newh - 1);
	float pom2 = (width - 1) / (neww - 1);


	for (int j = 0; j < steps; j++)
	{
		res = (thread*steps + j) * pom;
		h = (int)floor(res);
		if (h < 0) {
			h = 0;
		}
		else {
			if (h >= height - 1) {
				h = height - 2;
			}
		}
		u = res - h;
		for (int i = thread; i < neww; i++)
		{
			res = (i) * pom2;
			w = (int)floor(res);
			if (w < 0) {
				w = 0;
			}
			else {
				if (w >= width - 1) {
					w = width - 2;
				}
				t = res - w;

				/* ������������ */
				d1 = (1 - t) * (1 - u);
				d2 = t * (1 - u);
				d3 = t * u;
				d4 = (1 - t) * u;

				/* ��������� �������: a[i][j] */
				p1 = data1[w + h * width];
				p2 = data1[w + h * width + 1];
				p3 = data1[w + 1 + h * width + 1];
				p4 = data1[w + 1 + h * width];

				result_data[i + (thread*steps + j)* neww] = p1 * d1 + p2 * d2 + p3 * d3 + p4 * d4;
			}
		}
	}
}

int main()
{
	int blocks = 256;
	int blocksize = 512;
	if (!Targa2Array("C:/Users/B.Lokotkov/Desktop/_git/POVVS/Lab5/sample2.tga", &data1, &width, &height, &pbpp))
	{
		std::cout << "Can't read file";
		return -1;
	}
	int neww = (width - 1)*N + 1;
	int newh = (height - 1)*N + 1;
	int steps;
	float elapsedTime;
	hipEvent_t start, stop; //�������������� �������
	hipEventCreate(&start);
	hipEventCreate(&stop);

	unsigned char* dev_data;
	unsigned char* dev_result_data;
	unsigned char* result_data = new unsigned char[neww * newh];

	hipMalloc((void**)&dev_data, width * height * sizeof(unsigned char));
	hipMalloc((void**)&dev_result_data, neww * newh * sizeof(unsigned char));

	steps = (int)newh / (blocks*blocksize);

	hipEventRecord(start, 0); //�������� ������� start
	hipMemcpy(dev_data, data1, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(dev_result_data, result_data, height * width * sizeof(unsigned char), hipMemcpyHostToDevice);

	fun_kernel << < blocks, blocksize >> > (dev_result_data, dev_data, height, width, steps);
	hipMemcpy(result_data, dev_result_data, newh * neww * sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0); //�������� ������� stop
	hipEventSynchronize(stop); //������������� host � device �� ������� stop


	hipEventElapsedTime(&elapsedTime, start, stop);
	std::cout << "Time: " << elapsedTime;
	Array2Targa("result.tga", result_data, neww, newh, pbpp);
	hipFree(dev_data);
	hipFree(dev_result_data);
	_getch();
}


/*// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}*/
